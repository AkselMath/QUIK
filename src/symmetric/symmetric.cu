#include "hip/hip_runtime.h"
#include "int4.h"
#include "symmetric/symmetric_internal.h"
#include "util.h"

namespace QUIK::symmetric {
__global__ void quantizeCUDAKernel(Int4Storage *__restrict__ dst,
                                       const torch::Half *__restrict__ scale,
                                       const torch::Half *__restrict__ src,
                                       const unsigned rows,
                                       const unsigned colsSrc,
                                       unsigned colsDst) {
  const unsigned row = threadIdx.y + blockIdx.y * blockDim.y;
  const unsigned colDst = threadIdx.x + blockIdx.x * blockDim.x;
  if (row >= rows || colDst * kElementsPerVector >= colsSrc) {
    return;
  }
  Int4Storage storage;
  memset(&storage, 0, sizeof(storage));
  const unsigned id = colDst * kElementsPerVector + row * colsSrc;
#pragma unroll
  for (int i = 0; i < kElementsPerVector; ++i) {
    bool safe = (colDst * kElementsPerVector + i) < colsSrc;
    if (safe) {
      __half data = __hdiv(src[id + i], scale[row]);
      Int4Subbyte{reinterpret_cast<cutlass::int4b_t *>(&storage), i}.set(
          __half2int_rn(data));
    }
  }
  dst[colDst + row * colsDst] = storage;
}

torch::Tensor quantizeCUDA(const torch::Tensor &src,
                               const torch::Tensor &scale) {
  torch::checkSameGPU("quantize", {src, "src", 0}, {scale, "scale", 1});
  torch::checkSize("quantize", torch::TensorArg{scale, "scale", 1}, 0,
                   src.size(0));
  unsigned rows = src.size(0);
  unsigned colsSrc = src.size(1);
  unsigned colsDst = (colsSrc - 1) / kElementsPerVector + 1;
  auto dst =
      torch::empty({rows, colsDst},
                   torch::dtype(util::TorchDtypeDispatcher<Int4Storage>::value)
                       .device(src.device()));
  dim3 block{std::min<unsigned>(colsDst, 32), std::min<unsigned>(rows, 16)};
  dim3 grid{(colsDst - 1) / block.x + 1, (rows - 1) / block.y + 1};
  quantizeCUDAKernel<<<grid, block>>>(
      dst.data_ptr<Int4Storage>(), scale.data_ptr<torch::Half>(),
      src.data_ptr<torch::Half>(), rows, colsSrc, colsDst);
  return dst;
}

__global__ void dequantizationKernel(torch::Half *__restrict__ out,
                                     const int *__restrict__ x,
                                     const torch::Half *__restrict__ scaleRow,
                                     const torch::Half *__restrict__ scaleCol,
                                     const torch::Half *__restrict__ y,
                                     const unsigned rows, const unsigned cols) {
  const unsigned row = threadIdx.y + blockIdx.y * blockDim.y;
  const unsigned col = threadIdx.x + blockIdx.x * blockDim.x;
  if (col >= cols) {
    return;
  }

  if (row >= rows) {
    return;
  }

  __half xElement = __int2half_rn(x[col + row * cols]);

  out[col + row * cols] = __hfma(__hmul(xElement, scaleRow[row]), scaleCol[col],
                                 y[col + row * cols]);
}

torch::Tensor dequantizeCUDA(const torch::Tensor &x,
                                 const torch::Tensor &scaleRow,
                                 const torch::Tensor &scaleCol,
                                 const torch::Tensor &y) {
  torch::checkAllSameGPU("dequantize", {{x, "x", 0},
                                          {scaleRow, "scaleRow", 1},
                                          {scaleCol, "scaleCol", 2},
                                          {y, "y", 3}});
//  torch::checkSameNumel("dequantize", torch::TensorArg{x, "x", 0}, torch::TensorArg{y, "y", 1});
  unsigned rows = x.size(0);
  unsigned cols = x.size(1);
  torch::checkSize("dequantize", torch::TensorArg{scaleRow, "scaleRow", 1},
                   0, rows);
  torch::checkSize("dequantize", torch::TensorArg{scaleCol, "scaleCol", 2}, 0,
                   cols);
  auto out = torch::empty_like(y);
  dim3 block{std::min<unsigned>(cols, 16),
             std::min<unsigned>((rows - 1) + 1, 16)};
  dim3 grid{(cols - 1) / block.x + 1, (rows - 1) / block.y + 1};
  dequantizationKernel<<<grid, block>>>(
      out.data_ptr<torch::Half>(), x.data_ptr<int>(),
      scaleRow.data_ptr<torch::Half>(), scaleCol.data_ptr<torch::Half>(),
      y.data_ptr<torch::Half>(), rows, cols);
  return out;
}

}  // namespace QUIK::symmetric